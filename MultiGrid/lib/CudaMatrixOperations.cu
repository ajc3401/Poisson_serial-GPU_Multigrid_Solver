#include "CudaMatrixOperations.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include <cfloat>
#include "CudaMemoryHandler.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "CudaKernels.cu"
#include "hipsparse.h"

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(EXIT_FAILURE);                                                   \
    }                                                                          \
}

void matrixMultiply(const float* a, const float* b, float* c, size_t N, size_t M, size_t K)
{
	// We obtain the matrix (M X K ) = (M X N) * (N X K)
	hipblasHandle_t handle = NULL;
	hipblasCreate(&handle);

	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, N, &beta, c, M);

	hipDeviceSynchronize();
}

void matrixMultiply(const int* a, const int* b, int* c, size_t N, size_t M, size_t K)
{
	std::runtime_error("Integer matrix multiplication not currently supported.");
}
void matrixMultiply(const double* a, const double* b, double* c, size_t N, size_t M, size_t K)
{
	// We obtain the matrix (M X K ) = (M X N) * (N X K)
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const double alpha = 1.0;
	const double beta = 0.0;

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, N, &beta, c, M);

	hipDeviceSynchronize();

	std::cout << "Cublas Matrix Multiplication" << std::endl;
}

// Ax = y
void sparseMatrixVectorMultiply(void* Aval, void* Arow, void* Acol, void* x, void* y, size_t A_num_rows, size_t A_num_cols, size_t A_NNZ)
{
	float alpha = 1.0f;
	float beta = 0.0f;

	void* dBuffer {nullptr};
	size_t bufferSize = 0;

	hipsparseHandle_t handle = NULL;
	hipsparseSpMatDescr_t matA;
	hipsparseDnVecDescr_t vecX, vecY;
	
	CHECK_CUSPARSE(hipsparseCreate(&handle));

	CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_NNZ, Arow, Acol, Aval, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
	
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, A_num_cols, x, HIP_R_32F));
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, A_num_rows, y, HIP_R_32F));
	
	CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
		handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		&alpha, matA, vecX, &beta, vecY, HIP_R_32F,
		HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));

	HANDLE_ERROR( hipMalloc(&dBuffer, bufferSize) );

	CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		&alpha, matA, vecX, &beta, vecY, HIP_R_32F,
		HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

	HANDLE_ERROR(hipFree(dBuffer));

	CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
	CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
	CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
	CHECK_CUSPARSE(hipsparseDestroy(handle));

	hipDeviceSynchronize();
}

void sparseMatrixMatrixMultiply(void* Aval, void* Arow, void* Acol, void* Bval, void* Brow, void* Bcol,
	void* Cval, void* Crow, void* Ccol, size_t A_num_rows, size_t A_num_cols, size_t A_NNZ, 
	size_t B_num_rows, size_t B_num_cols, size_t B_NNZ, size_t& C_num_rows, size_t& C_num_cols, size_t& C_NNZ)
{
	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA, matB, matC;

	float alpha = 1.0f;
	float beta = 0.0f;

	void* dBuffer1 {nullptr}, * dBuffer2 {nullptr};
	size_t bufferSize1 = 0, bufferSize2 = 0;


	

	CHECK_CUSPARSE(hipsparseCreate(&handle));
		
	CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_NNZ,
		Arow, Acol, Aval,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
	CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_NNZ,
		Brow, Bcol, Bval,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
	CHECK_CUSPARSE(hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
		NULL, NULL, NULL,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

	hipsparseSpGEMMDescr_t spgemmDesc;
	CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc));


	CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				&alpha, matA, matB, &beta, matC,
				HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
				spgemmDesc, &bufferSize1, NULL));

	HANDLE_ERROR(hipMalloc((void**)&dBuffer1, bufferSize1));


	CHECK_CUSPARSE(
		hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, matA, matB, &beta, matC,
			HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
			spgemmDesc, &bufferSize1, dBuffer1));

	CHECK_CUSPARSE(
		hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, matA, matB, &beta, matC,
			HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
			spgemmDesc, &bufferSize2, NULL));
	
	HANDLE_ERROR(hipMalloc((void**)&dBuffer2, bufferSize2));

	CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		&alpha, matA, matB, &beta, matC,
		HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
		spgemmDesc, &bufferSize2, dBuffer2));
	
		int64_t C_num_rows1, C_num_cols1, C_nnz1;
	CHECK_CUSPARSE(hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
			&C_nnz1));
	// Set the number of rows, cols, NNZ
	C_num_rows = C_num_rows1;
	C_num_cols = C_num_cols1;
	C_NNZ = C_nnz1;

	// allocate matrix C
	

	CHECK_CUSPARSE(hipsparseCsrSetPointers(matC, Crow, Ccol, Cval));

	CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, matA, matB, &beta, matC,
			HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc));

	CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc));
	CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
	CHECK_CUSPARSE(hipsparseDestroySpMat(matB));
	CHECK_CUSPARSE(hipsparseDestroySpMat(matC));
	CHECK_CUSPARSE(hipsparseDestroy(handle));

	hipDeviceSynchronize();
	
}
void convertDensetoCSR(void* dense, size_t nrows, size_t ncols, void* csr_val, void* csr_row, void* csr_col, size_t& nNNZ)
{
	void* dBuffer {nullptr};
	size_t bufferSize = 0;
	size_t leading_dim{ nrows };
	
	hipsparseHandle_t handle = NULL;
	hipsparseSpMatDescr_t csr_mat;
	hipsparseDnMatDescr_t dense_mat;

	CHECK_CUSPARSE(hipsparseCreate(&handle));
	// Create dense matrix
	CHECK_CUSPARSE(hipsparseCreateDnMat(&dense_mat, nrows, ncols, leading_dim, dense,
		HIP_R_32F, HIPSPARSE_ORDER_COL));
	// Create sparse matrix
	CHECK_CUSPARSE(hipsparseCreateCsr(&csr_mat, nrows, ncols, 0,
		csr_row, NULL, NULL,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
	// Create external buffer
	CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, dense_mat, csr_mat, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
		&bufferSize))
		

	HANDLE_ERROR(hipMalloc(&dBuffer, bufferSize));

	// Do the actual dense to CSR conversion
	CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, dense_mat, csr_mat, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

	int64_t nrows_tmp, ncols_tmp, NNZ;
	CHECK_CUSPARSE(hipsparseSpMatGetSize(csr_mat, &nrows_tmp, &ncols_tmp, &NNZ));

	nNNZ = NNZ;

	CHECK_CUSPARSE(hipsparseCsrSetPointers(csr_mat, csr_row, csr_col, csr_val));

	CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, dense_mat, csr_mat, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));
		

	HANDLE_ERROR(hipFree(dBuffer));

	CHECK_CUSPARSE(hipsparseDestroyDnMat(dense_mat));
	CHECK_CUSPARSE(hipsparseDestroySpMat(csr_mat));
	CHECK_CUSPARSE(hipsparseDestroy(handle));

	hipDeviceSynchronize();

	
}

template <class T> void setDiagonal(T* a, const T number, size_t offset, size_t M, size_t N)
{
	size_t MATRIX_SIZE{ M * N };
	SetDiagonal << <(MATRIX_SIZE - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >> > (a, number, offset, M, N);
	hipDeviceSynchronize();
}

template <class T> void getDiagonal(T* a, const T* b, size_t offset, size_t M, size_t N)
{
	size_t MATRIX_SIZE{ M * N };
	GetDiagonal << <(MATRIX_SIZE - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >> > (a, b, offset, M, N);
	hipDeviceSynchronize();
}

template <class T> void getLowerTriangular(T* a, const T* b, size_t offset, size_t M, size_t N)
{
	
	size_t MATRIX_SIZE{ M * N };
	size_t GRID_SIZE{ (MATRIX_SIZE - 1) / BLOCK_SIZE + 1 };
	
	dim3 THREADS(BLOCK_SIZE, BLOCK_SIZE);
	dim3 GRID(GRID_SIZE, GRID_SIZE);
	
	GetLowerTriangular << <GRID, THREADS >> > (a, b, offset, M, N);
	hipDeviceSynchronize();
}

template <class T> void getUpperTriangular(T* a, const T* b, size_t offset, size_t M, size_t N)
{
	size_t MATRIX_SIZE{ M * N };

	size_t GRID_SIZE{ (MATRIX_SIZE - 1) / BLOCK_SIZE + 1 };

	dim3 THREADS(BLOCK_SIZE, BLOCK_SIZE);
	dim3 GRID(GRID_SIZE, GRID_SIZE);


	GetUpperTriangular << <GRID, THREADS >> > (a, b, offset, M, N);
	hipDeviceSynchronize();
	
}

template void setDiagonal(double* a, const double number, size_t offset, size_t M, size_t N);
template void setDiagonal(float* a, const float number, size_t offset, size_t M, size_t N);

template void getDiagonal(double* a, const double* b, size_t offset, size_t M, size_t N);
template void getDiagonal(float* a, const float* b, size_t offset, size_t M, size_t N);

template void getLowerTriangular(double* a, const double* b, size_t offset, size_t M, size_t N);
template void getLowerTriangular(float* a, const float* b, size_t offset, size_t M, size_t N);

template void getUpperTriangular(double* a, const double* b, size_t offset, size_t M, size_t N);
template void getUpperTriangular(float* a, const float* b, size_t offset, size_t M, size_t N);

