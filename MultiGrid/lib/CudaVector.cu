#include "hip/hip_runtime.h"
// Copyright 2022, Anthony Cooper, All rights reserved

#include "CudaVector.cuh"
#include "CudaVectorOperations.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include <cfloat>
#include "CudaMemoryHandler.cuh"

template <class T>
CudaVector<T>::CudaVector() : VectorBase<T>() 
{
	reserve(10);

}

template <class T>
CudaVector<T>::CudaVector(const size_t _nalloc) : VectorBase<T>	()
{
	reserve(_nalloc);
}

template<class T>
CudaVector<T>::CudaVector(std::initializer_list<T> init) : VectorBase<T>()
{
	
	resize(0, init.size());
	for (auto x : init)
	{
		this->push_back(x);
	}
	
}

template <class T>
CudaVector<T>::CudaVector(const CudaVector<T>& copy) : VectorBase<T> ()

{
	resize(copy.m_nelem, copy.m_nalloc);
	setEqual(this->m_ptr, copy.m_ptr, this->m_nelem);
	
}


template<class T>
CudaVector<T>::CudaVector(CudaVector<T>&& move) noexcept
{
	swap(*this, move);
}

template <class T>
CudaVector<T>::~CudaVector()
{
	for (size_t i = 0; i < m_nelem; i++)
	{
		m_ptr[m_nelem - 1 - i].~T();
	}
	CudaMemoryHandler<T>::deallocate(m_ptr);
}

template<class T>
CudaVector<T>& CudaVector<T>::operator=(const VectorBase<T>& rhs)
{
	assert(this->m_nelem == rhs.get_nelem());
	if (dynamic_cast<const CudaVector<T>*>(&rhs))
		*(this->m_ptr) = *(rhs.begin());
	else
		throw std::invalid_argument("Cannot equate a serial and GPU vector");
	
	return *this;
//	
}

template<class T>
CudaVector<T>& CudaVector<T>::operator=(const CudaVector<T>& rhs)
{
	CudaVector<T> tmp(rhs);
	swap(*this, tmp);
	return *this;
}

template<class T>
CudaVector<T>& CudaVector<T>::operator-()
{
	
	setNegative(this->m_ptr, this->m_nelem);
	return *this;
}

template <class T>
CudaVector<T>& CudaVector<T>::operator+=(const VectorBase<T>& rhs)
{
	assert(this->m_nelem == rhs.get_nelem());
	const CudaVector<T>* CudaVectorptr = dynamic_cast<const CudaVector<T>*>(&rhs);
	if (CudaVectorptr == nullptr)
		throw std::invalid_argument("Cannot add a serial vector to a GPU vector");
	else
		(*this) += rhs;

	return *this;
}

template <class T>
CudaVector<T>& CudaVector<T>::operator-=(const VectorBase<T>& rhs)
{
	assert(this->m_nelem == rhs.get_nelem());
	const CudaVector<T>* CudaVectorptr = dynamic_cast<const CudaVector<T>*>(&rhs);
	if (CudaVectorptr == nullptr)
		throw std::invalid_argument("Cannot add a serial vector to a GPU vector");
	else
		(*this) -= rhs;
	return *this;
}



template <class T>
CudaVector<T>& CudaVector<T>::operator+=(const CudaVector<T>& rhs)
{
	assert(this->m_nelem == rhs.m_nelem);
	sumVectors(this->m_ptr, rhs.m_ptr, this->m_nelem);

	return *this;
}

template <class T>
CudaVector<T>& CudaVector<T>::operator-=(const CudaVector<T>& rhs)
{
	assert(this->m_nelem == rhs.m_nelem);
	subtractVectors(this->m_ptr, rhs.m_ptr, this->m_nelem);

	return *this;
}

template <class T>
CudaVector<T>& CudaVector<T>::operator*=(const T& rhs)
{
	scalarVectorMultiply(this->m_ptr, rhs, this->m_nelem);
	return *this;
}

template <class T>
T CudaVector<T>::dot_product(const VectorBase<T>& rhs) const
{
	assert(this->m_nelem == rhs.get_nelem());
	const CudaVector<T>* CudaVectorptr = dynamic_cast<const CudaVector<T>*>(&rhs);
	if (CudaVectorptr == nullptr)
		throw std::invalid_argument("Cannot compute the dot product of a serial and GPU vector.");
	else
	{
		T c = 0;
		c = this->dot_product(rhs);
		return c;
	}
}

template <class T>
T CudaVector<T>::dot_product(const CudaVector<T>& rhs) const
{
	assert(this->m_nelem == rhs.m_nelem);
	T c{ 0 };
	const CudaVector<T> tmp(*this);
	dotProduct(tmp.m_ptr, rhs.m_ptr, c, this->m_nelem);
	return c;
}

template<class T>
void CudaVector<T>::invert_elements()
{
	invertElements(this->m_ptr, this->m_nelem);
}

template<class T>
void CudaVector<T>::interpolate_1D(const VectorBase<T>& v_coarser)
{
	const CudaVector<T>* CudaVectorptr = dynamic_cast<const CudaVector<T>*>(&v_coarser);
	if (CudaVectorptr == nullptr)
		throw std::invalid_argument("Cannot interpolate a serial vector to GPU vector");
	else
		this->interpolate_1D(*CudaVectorptr);
}

template<class T>
void CudaVector<T>::interject_1D(const VectorBase<T>& v_finer)
{
	const CudaVector<T>* CudaVectorptr = dynamic_cast<const CudaVector<T>*>(&v_finer);
	if (CudaVectorptr == nullptr)
		throw std::invalid_argument("Cannot interject a serial vector to GPU vector");
	else
		this->interject_1D(*CudaVectorptr);
}

template<class T>
T CudaVector<T>::l2norm()
{
	T* result = new T(0.0f);
	//const CudaVector<T> tmp(*this);
	//float* result = new float(0);
	//const CudaVector<float> tmp(*this);
	l2Norm(result, this->m_ptr, this->m_nelem);
	return *result;
}

template<class T>
void CudaVector<T>::sin()
{
	std::string s("sin");
	applyFunction(this->m_ptr, this->m_nelem, s);
}

template<class T>
void CudaVector<T>::interpolate_1D(const CudaVector<T>& v_coarser)
{
	const size_t N_finer{ 2 * this->m_nelem + 1 };
	const size_t N_coarser{ v_coarser.m_nelem };
	
	if(this->m_nelem != N_finer)
		this->resize(N_finer, N_finer);

	interpolate1D(this->m_ptr, v_coarser.m_ptr, N_coarser);
	
	this->m_ptr[0] = 0.5 * v_coarser.m_ptr[0];
	this->m_ptr[N_finer - 1] = 0.5 * v_coarser.m_ptr[N_coarser - 1];
}

template<class T>
void CudaVector<T>::interject_1D(const CudaVector<T>& v_finer)
{
	const size_t N_finer{ v_finer.m_nelem };
	const size_t N_coarser{ static_cast<size_t>(0.5*(N_finer -1) + 1)};
	std::cout << "N_coarser = " << N_coarser << std::endl;
	if(this->m_nelem != N_coarser)
		this->resize(N_coarser, N_coarser);

	this->m_ptr[0] = v_finer.m_ptr[0];
	std::cout << v_finer.m_ptr[N_finer - 1] << std::endl;
	interject1D(this->m_ptr, v_finer.m_ptr, N_coarser);
	
	this->m_ptr[N_coarser - 1] = v_finer.m_ptr[N_finer - 1];
	
}

template<class T>
void CudaVector<T>::set_to_number(const T number)
{
	//CudaVector<T> tmp(*this);
	setValue(this->m_ptr, number, this->m_nelem);
	//for (size_t i = 0; i < tmp.m_nelem; i++)
	//	this->m_ptr[i] = tmp.m_ptr[i];
}

template<class T>
void CudaVector<T>::set_to_range(size_t left, size_t right, const VectorBase<T>& invec)
{
	const CudaVector<T>* CudaVectorptr = dynamic_cast<const CudaVector<T>*>(&invec);
	if (CudaVectorptr == nullptr)
		throw std::invalid_argument("Cannot set elements of a GPU vector to a serial vector.");
	else
		this->set_to_range(left, right, *CudaVectorptr);
}

template<class T>
void CudaVector<T>::set_to_range(size_t left, size_t right, const CudaVector<T>& invec)
{
	size_t rangelength = right - left;
	if (this->m_nelem != rangelength)
		this->resize(rangelength, rangelength);
	setRange(left, right, this->m_ptr, invec.m_ptr);
}

template<class T>
void CudaVector<T>::push_back(const T& element)
{
	if (m_nelem < m_nalloc) {

		m_ptr[m_nelem] = element;
		++m_nelem;
	}
	else
	{
		this->resize(m_nalloc * 1.5);
		new (m_ptr + m_nelem) T(element);
		++m_nelem;
	}

}


template<class T>
void CudaVector<T>::push_back(T&& element)
{
	if (m_nelem < m_nalloc) {

		m_ptr[m_nelem] = std::move(element);
		++m_nelem;
	}
	else
	{
		this->resize(m_nalloc * 1.5);
		new (m_ptr + m_nelem) T(std::move(element));
		++m_nelem;
	}

}

template<class T>
template<typename... Args> T& CudaVector<T>::emplace_back(Args&&... args)
{
	if (this->m_nelem < this->m_nalloc) {
		this->m_ptr[this->m_nelem] = T(std::forward<Args>(args)...);
		++this->m_nelem;
	}
	else
	{
		this->resize(this->m_nalloc * 1.5);
		this->m_ptr[this->m_nelem] = T(std::forward<Args>(args)...);
		++this->m_nelem;
	}
	return this->m_ptr[this->m_nelem];
}

template<class T>
void CudaVector<T>::reserve(const size_t _nalloc)
{
	assert(_nalloc > m_nelem);
	
	CudaMemoryHandler<T>::deallocate(this->m_ptr);
	this->m_ptr = nullptr;
	this->m_nalloc = this->m_nelem = 0;

	this->m_nalloc = _nalloc;

	this->m_ptr = CudaMemoryHandler<T>::allocate(_nalloc);
	
}

template<class T>
void CudaVector<T>::resize(const size_t _nelem, const size_t _nalloc)
{
	// We only allow growing, not shrinking
	assert((_nelem >= m_nelem) && (_nalloc >= m_nelem));

	CudaVector<T> tmp(_nalloc);
	if (m_nelem > 0)
	{
		for (auto& x : *this)
			tmp.emplace_back(x);
	}
	for (size_t i = m_nelem; i < _nelem; i++)
		tmp.emplace_back(0);
	

	swap(tmp, *this);
	
}

template<class T>
void CudaVector<T>::resize(const size_t _nalloc)
{
	
	assert(_nalloc > this->m_nelem);

	CudaVector<T> tmp(_nalloc);
	if (m_nelem > 0)
	{
		for (auto& x : *this)
			tmp.emplace_back(x);
	}


	swap(tmp, *this);

}

template<class T>
void CudaVector<T>::pop_back()
{
	if (m_nelem > 0)
	{
		this->m_nelem--;
		this->m_ptr[this->m_nelem].~T();
	}

}

template<class T>
void CudaVector<T>::swap(VectorBase<T>& a, VectorBase<T>& b)
{
	CudaVector<T>* CudaVectorptra = dynamic_cast<CudaVector<T>*>(&a);
	CudaVector<T>* CudaVectorptrb = dynamic_cast<CudaVector<T>*>(&b);
	if (CudaVectorptra == nullptr || CudaVectorptrb == nullptr)
		throw std::invalid_argument("Cannot swap a serial and GPU vector");
	else
		swap(a, b);
}


template<class T>
void CudaVector<T>::swap(CudaVector<T>& a, CudaVector<T>& b)
{

	std::swap(a.m_nalloc, b.m_nalloc);
	std::swap(a.m_nelem, b.m_nelem);
	std::swap(a.m_ptr, b.m_ptr);
}

template<class T>
void CudaVector<T>::display() const
{
	for (size_t i = 0; i < m_nelem; i++)
	{
		std::cout << this->m_ptr[i] << std::endl;
	}
}

template class CudaVector<double>;
template class CudaVector<float>;
template class CudaVector<size_t>;
template class CudaVector<int>;

